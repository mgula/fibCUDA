#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define CHUNK_SIZE 1024
#define T unsigned long int

//make sure start is less than N/2. a is a pointer to an array of length >= N
__global__ void Fibonacci( T *a, int start) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int index = i + start;
	if (i < 2 * start - 1) {
		a[index] = (a[start - 2] * a[i]) + (a[start - 1] * a[i + 1]);
	}   
}

int main(int argc, char *argv[]) {
	int N = 0;

	if (argc == 2) {
		N = atoi(argv[1]);
	} else {
		printf("Invalid number of command line arguments.\n");
		return 1;
	}
	
	T x[3]= {1, 1, 2};
	T *d_a;

	//Allocate memory on the device
	hipMalloc(&d_a, N*sizeof(T));
	hipMemcpy(d_a, x, sizeof(x), hipMemcpyHostToDevice);

	unsigned int start = 3;

	//ceiling of start - 1/1024
	while (start <= N/2 ) {
		unsigned int num_blocks = (start - 1)/CHUNK_SIZE;
		if ((start - 1) % CHUNK_SIZE != 0) {
			num_blocks++;
		}
		Fibonacci <<<num_blocks, CHUNK_SIZE>>>(d_a, start);
		start = 2 * start - 1;
	}

	//Get the results array back
	T b[N];
	hipMemcpy(b, d_a, N*sizeof(T), hipMemcpyDeviceToHost);

	//Print results
	for (int i = 0; i < N; i++) {
		printf("%d:\t%lu \n", i + 1, b[i]);
	}
	
	//Free device memory
	hipFree(d_a);

	fflush(stdout);
	return 0;
}
